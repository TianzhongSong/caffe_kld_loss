#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/kld_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void KLD_fwd_gpu(const int N,
          const Dtype* p, const Dtype* q, Dtype* loss) {
  CUDA_KERNEL_LOOP(index, N) {
    loss[index] = log(max(p[index],Dtype(FLT_MIN)));
    loss[index] -= log(max(q[index],Dtype(FLT_MIN)));

  }
}

template <typename Dtype>
void KLDLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* prob_data = bottom[0]->gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  Dtype* temp = bottom[0]->mutable_gpu_diff();

  int N = bottom[0]->count();
  Dtype loss = 0;
  
  KLD_fwd_gpu<Dtype><<<CAFFE_GET_BLOCKS(N),CAFFE_CUDA_NUM_THREADS>>>(
     N,prob_data,label,temp);
  caffe_gpu_dot(N,temp,label,&loss);

  top[0]->mutable_cpu_data()[0] = -loss / 
      get_normalizer(normalization_, Dtype(N));
}

template <typename Dtype>
void KLDLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to target distribution yet.";
  }

  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = bottom[0]->gpu_data();
    const Dtype* label = bottom[1]->gpu_data();
    int N = bottom[0]->count();
    
    caffe_gpu_sub(N,prob_data,label,bottom_diff);

    Dtype loss_weight = top[0]->cpu_diff()[0] /
      get_normalizer(normalization_, Dtype(N));

    caffe_gpu_scal(N, loss_weight , bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(KLDLossLayer);

}  // namespace caffe